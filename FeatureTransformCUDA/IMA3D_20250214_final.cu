#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <string>
#include <queue>
#include <tuple>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "volume.h"
#include "loadTiff.h"
#include "TimerClock.hpp"
#include "compaction.h"
#include "fastmarching.h"
#include "threshold.h"

#include <hip/hip_runtime.h> 
#include <>

const int GAMMA = 1;
const int FOREGROUND = 1;
const int BACKGROUND = 0;
const int SKEL = 255;

#define sqr(x) ((x)*(x))
#define sqrlength(i, j, k) ((i)*(i)+(j)*(j)+(k)*(k))

static int			gamma_val = GAMMA;
static IntVolume*	ft,*ft_test;
static ByteVolume*	indata, *indata_test;
static char			input_file[MAXSTR];
static char			output_file[MAXSTR];
static char*		basefilename;
static char			basename[MAXSTR];
static char			skel_file[MAXSTR];

__global__ void skeletonKernel(BYTE* skel, BYTE* dis, int xdim, int ydim, int zdim) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;

	//if (x <= 0 || x >= xdim - 1 || y <= 0 || y >= ydim - 1 || z <= 0 || z >= zdim - 1)
	//	return;

	int idx = x + y * xdim + z * xdim * ydim;

	if (dis[idx] == 0)
	{
		skel[idx] = 0;
		return;
	}
		

	int neighborOffsets[26][3] = {
		{-1, -1, -1}, { 0, -1, -1}, { 1, -1, -1},
		{-1,  0, -1}, { 0,  0, -1}, { 1,  0, -1},
		{-1,  1, -1}, { 0,  1, -1}, { 1,  1, -1},

		{-1, -1,  0}, { 0, -1,  0}, { 1, -1,  0},
		{-1,  0,  0},               { 1,  0,  0},
		{-1,  1,  0}, { 0,  1,  0}, { 1,  1,  0},

		{-1, -1,  1}, { 0, -1,  1}, { 1, -1,  1},
		{-1,  0,  1}, { 0,  0,  1}, { 1,  0,  1},
		{-1,  1,  1}, { 0,  1,  1}, { 1,  1,  1},
	};

	BYTE maxValues[8] = { 0 };

	for (int i = 0; i < 26; i++) {
		int nx = x + neighborOffsets[i][0];
		int ny = y + neighborOffsets[i][1];
		int nz = z + neighborOffsets[i][2];

		if (nx >= 0 && nx < xdim && ny >= 0 && ny < ydim && nz >= 0 && nz < zdim) {
			int neighborIdx = nx + ny * xdim + nz * xdim * ydim;
			BYTE neighborValue = dis[neighborIdx];

			int direction = ((nx >= x) << 2) | ((ny >= y) << 1) | (nz >= z);
			maxValues[direction] = fmaxf(maxValues[direction], neighborValue);
		}
	}

	// 这里不一定只要有一个最大值大于当前点的值，就可以认为当前点是骨架点，可以改成一半或者几个
	for (int i = 0; i < 8; i++) {
		if (dis[idx] > maxValues[i]) {
			skel[idx] = SKEL;
			return;
		}
	}
	skel[idx] = 0;
}

__global__ void skeletonKernel_compact(unsigned char* skel_compact, unsigned char* dis_compact, int* d_compress, int* d_decompress, int width, int height, int slice, int newSize) {
	
	int smallIdx = blockDim.x * blockIdx.x + threadIdx.x;
	if (smallIdx >= newSize) return;

	int fullIdx = d_decompress[smallIdx];
	if (fullIdx == -1) return;

	int z = fullIdx / (width * height);
	int y = fullIdx % (width * height) / width;
	int x = fullIdx % width;

	if (dis_compact[smallIdx] == 0)
	{
		skel_compact[smallIdx] = 0;
		return;
	}

	unsigned char curDistValue = dis_compact[smallIdx];


	int neighborOffsets[26][3] = {
		{-1, -1, -1}, { 0, -1, -1}, { 1, -1, -1},
		{-1,  0, -1}, { 0,  0, -1}, { 1,  0, -1},
		{-1,  1, -1}, { 0,  1, -1}, { 1,  1, -1},

		{-1, -1,  0}, { 0, -1,  0}, { 1, -1,  0},
		{-1,  0,  0},               { 1,  0,  0},
		{-1,  1,  0}, { 0,  1,  0}, { 1,  1,  0},

		{-1, -1,  1}, { 0, -1,  1}, { 1, -1,  1},
		{-1,  0,  1}, { 0,  0,  1}, { 1,  0,  1},
		{-1,  1,  1}, { 0,  1,  1}, { 1,  1,  1},
	};

	BYTE maxValues[8] = { 0 };

	for (int i = 0; i < 26; i++) {
		int nx = x + neighborOffsets[i][0];
		int ny = y + neighborOffsets[i][1];
		int nz = z + neighborOffsets[i][2];

		if (nx >= 0 && nx < width && ny >= 0 && ny < height && nz >= 0 && nz < slice) {
			int neighborFullIdx = nz * width * height + ny * width + nx;
			int neighborSmallIdx = d_compress[neighborFullIdx];
			if (neighborSmallIdx == -1) continue;

			unsigned char neighborValue = dis_compact[neighborSmallIdx];

			int direction = ((nx >= x) << 2) | ((ny >= y) << 1) | (nz >= z);
			maxValues[direction] = fmaxf(maxValues[direction], neighborValue);
		}
	}

	// 这里不一定只要有一个最大值大于当前点的值，就可以认为当前点是骨架点，可以改成一半或者几个
	for (int i = 0; i < 8; i++) {
		if (curDistValue > maxValues[i]) {
			skel_compact[smallIdx] = SKEL;
			return;
		}
	}
	skel_compact[smallIdx] = 0;
}

__global__ void processArrayKernel(unsigned char* data, int size, int FOREGROUND, int SKEL) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) {
		if (data[idx] == FOREGROUND) {
			data[idx] = 0;
			printf("modified!\n");
		}
		// 就是255不用动
		/*if (data[idx] == SKEL) {
			data[idx] = 1;
		}*/
	}
}

__global__ void meanBlurKernel(BYTE* input, BYTE* output, int xdim, int ydim, int zdim, int kernelRadius) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x < kernelRadius || x >= xdim - kernelRadius ||
		y < kernelRadius || y >= ydim - kernelRadius ||
		z < kernelRadius || z >= zdim - kernelRadius) {
		return;
	}

	int idx = x + y * xdim + z * xdim * ydim;

	float sum = 0.0f;
	int count = 0;

	for (int dz = -kernelRadius; dz <= kernelRadius; dz++) {
		for (int dy = -kernelRadius; dy <= kernelRadius; dy++) {
			for (int dx = -kernelRadius; dx <= kernelRadius; dx++) {
				int nx = x + dx;
				int ny = y + dy;
				int nz = z + dz;

				int neighborIdx = nx + ny * xdim + nz * xdim * ydim;

				sum += input[neighborIdx];
				count++;
			}
		}
	}

	output[idx] = (BYTE)(sum / count);
}

template <typename T>
__global__
void compare(T* arr1, T* arr2, int arrSize)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= arrSize) return;
	if (arr1[idx] != arr2[idx])
		printf("mismatch: id: %d,  arr1: %d, arr2: %d\n", idx, (int)(arr1[idx]), (int)(arr2[idx]));
}

void GetSkeleton(unsigned char* ImagePtr, int xdim, int ydim, int zdim)
{
	TimerClock timer;

	unsigned char* d_imagePtr_ori = nullptr;
	hipMalloc(&d_imagePtr_ori, sizeof(unsigned char) * xdim * ydim * zdim);
	hipMemcpy(d_imagePtr_ori, ImagePtr, sizeof(unsigned char) * xdim * ydim * zdim, hipMemcpyHostToDevice);

	unsigned char* d_imagePtr_cpy = nullptr;
	hipMalloc(&d_imagePtr_cpy, sizeof(unsigned char) * xdim * ydim * zdim);
	hipMemcpy(d_imagePtr_cpy, ImagePtr, sizeof(unsigned char) * xdim * ydim * zdim, hipMemcpyHostToDevice);

	int globalThreshold = 3;
	globalThreshold = 3; //大图
	//globalThreshold = 70; //小图
	addGlobalThreshold(d_imagePtr_ori, xdim, ydim, zdim, globalThreshold);

	unsigned char* d_imagePtr = nullptr;
	hipMalloc(&d_imagePtr, sizeof(unsigned char) * xdim * ydim * zdim);

	dim3 blockSize1(8, 8, 8);
	dim3 gridSize1((xdim + blockSize1.x - 1) / blockSize1.x,
		(ydim + blockSize1.y - 1) / blockSize1.y,
		(zdim + blockSize1.z - 1) / blockSize1.z);
	//meanBlurKernel << <gridSize1, blockSize1 >> > (d_imagePtr_ori, d_imagePtr, xdim, ydim, zdim, 0); //大图
	meanBlurKernel << <gridSize1, blockSize1 >> > (d_imagePtr_ori, d_imagePtr, xdim, ydim, zdim, 3); //小图


	addMaxMinGlobalThreshold(d_imagePtr_cpy, xdim, ydim, zdim, globalThreshold);
	std::cerr << "set globalThreshold cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	//将原图中紧靠前景点的背景点赋值为1，作为之后的扩展起点
	addDarkPadding(d_imagePtr, xdim, ydim, zdim, globalThreshold);
	std::cerr << "add darkpadding cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	int* d_compress; //压缩映射矩阵
	int* d_decompress; //解压缩映射矩阵
	unsigned char* d_imagePtr_compact; //压缩后原图
	int newSize; //压缩后数组总大小

	compactImage(d_imagePtr, d_imagePtr_compact, d_compress, d_decompress, xdim, ydim, zdim, newSize);


	unsigned char* d_imagePtr_compact_copy;
	hipMalloc(&d_imagePtr_compact_copy, sizeof(unsigned char) * newSize);
	hipMemcpy(d_imagePtr_compact_copy, d_imagePtr_compact, sizeof(unsigned char) * newSize, hipMemcpyDeviceToDevice);


	std::cerr << "OldSize: " << xdim * ydim * zdim << " NewSize: " << newSize << std::endl;
	printf("Compress Ratio: %.2lf%%\n", 100.0 * newSize / (1.0 * xdim * ydim * zdim));

	hipDeviceSynchronize();
	std::cerr << "compaction cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	//测试1: 直接将原图变换为距离值，调用addGreyWeightTransform() 函数 将d_imagePtr_compact 变为距离变换之后的数组

	addGreyWeightTransform(d_imagePtr, d_imagePtr_compact, d_compress, d_decompress, xdim, ydim, zdim, newSize);
	hipDeviceSynchronize();
	std::cerr << "GreyWeight Transform cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	//调用recoverImage() 将d_imagePtr_compact中的数据解压缩到原图d_imagePtr中
	recoverImage(d_imagePtr, d_imagePtr_compact, d_decompress, newSize);

	hipDeviceSynchronize();
	timer.update();

	int width = xdim; int height = ydim; int slice = zdim;
	int blockSize_compact = 256;
	int numblocks_compact = (newSize + 256 - 1) / 256;
	skeletonKernel_compact << <numblocks_compact, blockSize_compact >> > (d_imagePtr_compact_copy, d_imagePtr_compact, d_compress, d_decompress, width, height, slice, newSize);
	
	hipDeviceSynchronize();
	std::cerr << "skeletonKernel_compact cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();


	skeletonKernel << <gridSize1, blockSize1 >> > (d_imagePtr_cpy, d_imagePtr, xdim, ydim, zdim);

	hipDeviceSynchronize();
	std::cerr << "skeletonKernel cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	int blockSize2 = 256;
	int numBlocks = (xdim * ydim * zdim + 256 - 1) / 256;

	unsigned char* d_imagePtr_cpy_2;
	hipMalloc(&d_imagePtr_cpy_2, sizeof(unsigned char) * width * height * slice);
	recoverImage(d_imagePtr_cpy_2, d_imagePtr_compact_copy, d_decompress, newSize);


	//processArrayKernel << <numBlocks, blockSize2 >> > (d_imagePtr_cpy, xdim * ydim * zdim, FOREGROUND, SKEL);

	compare<unsigned char> << <numBlocks, blockSize2 >> > (d_imagePtr_cpy, d_imagePtr_cpy_2, width * height * slice);


	indata = ByteVolume_New(xdim, ydim, zdim);
	hipMemcpy(indata->data, d_imagePtr_cpy, xdim * ydim * zdim * sizeof(BYTE), hipMemcpyDeviceToHost);
	
	
	
	hipFree(d_imagePtr);
	hipFree(d_imagePtr_cpy);
	hipFree(d_compress);
	hipFree(d_decompress);
	hipFree(d_imagePtr_compact);
	hipFree(d_imagePtr_compact_copy);
	hipFree(d_imagePtr_cpy_2);
}

/*************** MAIN PROGRAM *****************/


void findConnectedRegion6(unsigned char* h_imagePtr, unsigned char* h_output,
	int width, int height, int slices,
	int xpos, int ypos, int zpos) {
	// 检查起始点是否在有效范围内
	if (xpos < 0 || xpos >= width || ypos < 0 || ypos >= height || zpos < 0 || zpos >= slices) {
		return;
	}

	// 计算起始点的线性索引
	const size_t start_index = zpos * width * height + ypos * width + xpos;
	if (h_imagePtr[start_index] == 0) {
		return; // 起始点不可联通
	}

	// 初始化队列和标记起始点
	std::queue<std::tuple<int, int, int>> bfs_queue;
	bfs_queue.push(std::make_tuple(xpos, ypos, zpos));
	h_output[start_index] = 255;

	// 6个邻域方向：±x, ±y, ±z
	const int directions[6][3] = {
		{1, 0, 0}, {-1, 0, 0},
		{0, 1, 0}, {0, -1, 0},
		{0, 0, 1}, {0, 0, -1}
	};

	while (!bfs_queue.empty()) {
		auto current = bfs_queue.front();
		bfs_queue.pop();

		int x = std::get<0>(current);
		int y = std::get<1>(current);
		int z = std::get<2>(current);

		for (const auto& dir : directions) {
			int nx = x + dir[0];
			int ny = y + dir[1];
			int nz = z + dir[2];

			// 边界检查
			if (nx >= 0 && nx < width && ny >= 0 && ny < height && nz >= 0 && nz < slices) {
				const size_t neighbor_index = nz * width * height + ny * width + nx;

				// 检查是否联通且未被访问
				if (h_imagePtr[neighbor_index] != 0 && h_output[neighbor_index] == 0) {
					h_output[neighbor_index] = 255;
					bfs_queue.push(std::make_tuple(nx, ny, nz));
				}
			}
		}
	}
}



int main(int argc, const char **argv)
{
	int i;
	int infile = -1;
	int outfile = -1;
	int gammavalue = -1;
	int xdim, ydim, zdim;
	BYTE max;

	TimerClock timer;

	{


		/* Parse command line params */
		for (i = 1; i < argc; i++)
		{
			if (strcmp(argv[i], "--help") == 0) {
				printf("\nUsage: %s INFILE [-g gamma] -o OUTFILE\n", argv[0]);
				printf("INFILE is the VTK file (unsigned char ()) to use as input.\n");
				printf("gamma is a value for the pruning parameter (default=1)\n");
				printf("gamma>1: constant pruning; gamma<1: linear pruning; gamma=0: square-root pruning.\n");
				return 0;
			}
			else if (strcmp(argv[i], "-g") == 0) {
				if (i + 1 < argc) {
					gammavalue = i + 1;
					i++;
				}
				else printf("Missing value for gamma.\n");
			}
			else if (strcmp(argv[i], "-o") == 0) {
				if (i + 1 < argc) {
					outfile = i + 1;
					i++;
				}
				else printf("Missing value for output file name.\n");
			}
			else infile = i;
		}

		if (infile == -1) {
			printf("Missing input file_name. Use '%s --help' for full help.\n", argv[0]);
			return 0;
		}
		if (outfile == -1) {
			printf("Missing output file_name. Use '%s --help' for full help.\n", argv[0]);
			return 0;
		}

		if (gammavalue != -1)
			gamma_val = atof(argv[gammavalue]);

		strcpy(input_file, argv[infile]);

		strcpy(output_file, argv[outfile]);

		basefilename = basename_no_ext(input_file, "vtk");

		sprintf(skel_file, "%s_%s%d_%s%s", basefilename, "g=", gamma_val, "skel", ".vtk");
		fprintf(stdout, "gamma = %d\n", gamma_val);
		fflush(stdout);

	}

	std::cerr << "Begin " << std::endl << std::endl;
	timer.update();

	auto size = new int[3];
	unsigned char* h_inputImagePtr = loadImage(input_file, size);
	xdim = size[0];
	ydim = size[1];
	zdim = size[2];
	int allsize = xdim * ydim * zdim;

	std::cerr << "Load cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	GetSkeleton(h_inputImagePtr, xdim, ydim, zdim);

	std::cerr << "Feature Trans cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	saveTiff(output_file, indata->data, size);


	
	int xpos = 133;
	int ypos = 341;
	int zpos = 45;

	unsigned char* h_connected = (unsigned char*)malloc(sizeof(unsigned char) * xdim * ydim * zdim);
	findConnectedRegion6(indata->data, h_connected, xdim, ydim, zdim, xpos, ydim - ypos - 1, zpos);

	saveTiff("connected.tif", h_connected, size);


	free(h_connected);
	// Cleanup
	ByteVolume_Delete(indata);

	return 0;
}
