#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <string>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "volume.h"
#include "loadTiff.h"
#include "TimerClock.hpp"
#include "compaction.h"
#include "fastmarching.h"
#include "threshold.h"

#include <hip/hip_runtime.h> 
#include <>

const int GAMMA = 1;
const int FOREGROUND = 1;
const int BACKGROUND = 0;
const int SKEL = 255;

#define sqr(x) ((x)*(x))
#define sqrlength(i, j, k) ((i)*(i)+(j)*(j)+(k)*(k))

static int			gamma_val = GAMMA;
static IntVolume*	ft,*ft_test;
static ByteVolume*	indata, *indata_test;
static char			input_file[MAXSTR];
static char			output_file[MAXSTR];
static char*		basefilename;
static char			basename[MAXSTR];
static char			skel_file[MAXSTR];

__global__ void skeletonKernel(BYTE* skel, BYTE* dis, int xdim, int ydim, int zdim) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x <= 0 || x >= xdim - 1 || y <= 0 || y >= ydim - 1 || z <= 0 || z >= zdim - 1)
		return;

	int idx = x + y * xdim + z * xdim * ydim;

	if (dis[idx] == 0)
		return;

	int neighborOffsets[26][3] = {
		{-1, -1, -1}, { 0, -1, -1}, { 1, -1, -1},
		{-1,  0, -1}, { 0,  0, -1}, { 1,  0, -1},
		{-1,  1, -1}, { 0,  1, -1}, { 1,  1, -1},

		{-1, -1,  0}, { 0, -1,  0}, { 1, -1,  0},
		{-1,  0,  0},               { 1,  0,  0},
		{-1,  1,  0}, { 0,  1,  0}, { 1,  1,  0},

		{-1, -1,  1}, { 0, -1,  1}, { 1, -1,  1},
		{-1,  0,  1}, { 0,  0,  1}, { 1,  0,  1},
		{-1,  1,  1}, { 0,  1,  1}, { 1,  1,  1},
	};

	BYTE maxNeighborValue = 0;
	for (int i = 0; i < 26; i++) {
		int nx = x + neighborOffsets[i][0];
		int ny = y + neighborOffsets[i][1];
		int nz = z + neighborOffsets[i][2];

		if (nx >= 0 && nx < xdim && ny >= 0 && ny < ydim && nz >= 0 && nz < zdim) {
			if (nx <= x && ny <= y && nz <= z) {
				int neighborIdx = nx + ny * xdim + nz * xdim * ydim;
				maxNeighborValue = fmaxf(maxNeighborValue, dis[neighborIdx]);
			}
		}
	}

	if (maxNeighborValue > 0)
		skel[idx] = SKEL;
	/*if (dis[idx] < maxNeighborValue) {
		skel[idx] = SKEL;
	}*/
}

__global__ void processArrayKernel(unsigned char* data, int size, int FOREGROUND, int SKEL) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) {
		if (data[idx] == FOREGROUND) {
			data[idx] = 0;
		}
		if (data[idx] == SKEL) {
			data[idx] = 1;
		}
	}
}

__global__ void meanBlurKernel(BYTE* input, BYTE* output, int xdim, int ydim, int zdim, int kernelRadius) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x < kernelRadius || x >= xdim - kernelRadius ||
		y < kernelRadius || y >= ydim - kernelRadius ||
		z < kernelRadius || z >= zdim - kernelRadius) {
		return;
	}

	int idx = x + y * xdim + z * xdim * ydim;

	float sum = 0.0f;
	int count = 0;

	for (int dz = -kernelRadius; dz <= kernelRadius; dz++) {
		for (int dy = -kernelRadius; dy <= kernelRadius; dy++) {
			for (int dx = -kernelRadius; dx <= kernelRadius; dx++) {
				int nx = x + dx;
				int ny = y + dy;
				int nz = z + dz;

				int neighborIdx = nx + ny * xdim + nz * xdim * ydim;

				sum += input[neighborIdx];
				count++;
			}
		}
	}

	output[idx] = (BYTE)(sum / count);
}

void GetSkeleton(unsigned char* ImagePtr, int xdim, int ydim, int zdim)
{
	TimerClock timer;

	unsigned char* d_imagePtr_ori = nullptr;
	hipMalloc(&d_imagePtr_ori, sizeof(unsigned char) * xdim * ydim * zdim);
	hipMemcpy(d_imagePtr_ori, ImagePtr, sizeof(unsigned char) * xdim * ydim * zdim, hipMemcpyHostToDevice);

	unsigned char* d_imagePtr_cpy = nullptr;
	hipMalloc(&d_imagePtr_cpy, sizeof(unsigned char) * xdim * ydim * zdim);
	hipMemcpy(d_imagePtr_cpy, ImagePtr, sizeof(unsigned char) * xdim * ydim * zdim, hipMemcpyHostToDevice);

	int globalThreshold = 60;
	globalThreshold = 2;
	addGlobalThreshold(d_imagePtr_ori, xdim, ydim, zdim, globalThreshold);

	//将原图中紧靠前景点的背景点赋值为1，作为之后的扩展起点
	addDarkPadding(d_imagePtr_ori, xdim, ydim, zdim, globalThreshold);
	std::cerr << "add darkpadding cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	unsigned char* d_imagePtr = nullptr;
	hipMalloc(&d_imagePtr, sizeof(unsigned char) * xdim * ydim * zdim);

	

	dim3 blockSize1(8, 8, 8);
	dim3 gridSize1((xdim + blockSize1.x - 1) / blockSize1.x,
		(ydim + blockSize1.y - 1) / blockSize1.y,
		(zdim + blockSize1.z - 1) / blockSize1.z);
	meanBlurKernel << <gridSize1, blockSize1 >> > (d_imagePtr_ori, d_imagePtr, xdim, ydim, zdim, 0);

	unsigned char* d_imagePtr_host = (unsigned char*)malloc(sizeof(unsigned char) * xdim * ydim * zdim);
	hipMemcpy(d_imagePtr_host, d_imagePtr, sizeof(unsigned char) * xdim * ydim * zdim, hipMemcpyDeviceToHost);

	addMaxMinGlobalThreshold(d_imagePtr_cpy, xdim, ydim, zdim, globalThreshold);
	std::cerr << "set globalThreshold cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	

	int* d_compress; //压缩映射矩阵
	int* d_decompress; //解压缩映射矩阵
	unsigned char* d_imagePtr_compact; //压缩后原图
	int newSize; //压缩后数组总大小

	compactImage(d_imagePtr, d_imagePtr_compact, d_compress, d_decompress, xdim, ydim, zdim, newSize);

	std::cerr << "OldSize: " << xdim * ydim * zdim << " NewSize: " << newSize << std::endl;
	printf("Compress Ratio: %.2lf%%\n", 100.0 * newSize / (1.0 * xdim * ydim * zdim));

	hipDeviceSynchronize();
	std::cerr << "compaction cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	//测试1: 直接将原图变换为距离值，调用addGreyWeightTransform() 函数 将d_imagePtr_compact 变为距离变换之后的数组

	addGreyWeightTransform(d_imagePtr, d_imagePtr_compact, d_compress, d_decompress, xdim, ydim, zdim, newSize);
	hipDeviceSynchronize();
	std::cerr << "GreyWeight Transform cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	//调用recoverImage() 将d_imagePtr_compact中的数据解压缩到原图d_imagePtr中
	recoverImage(d_imagePtr, d_imagePtr_compact, d_decompress, newSize);

	hipMemset(d_imagePtr_cpy, 0, sizeof(unsigned char) * xdim * ydim * zdim);

	skeletonKernel << <gridSize1, blockSize1 >> > (d_imagePtr_cpy, d_imagePtr, xdim, ydim, zdim);

	int blockSize2 = 256;
	int numBlocks = (xdim * ydim * zdim + 256 - 1) / 256;

	processArrayKernel << <numBlocks, blockSize2 >> > (d_imagePtr_cpy, xdim * ydim * zdim, FOREGROUND, SKEL);

	indata = ByteVolume_New(xdim, ydim, zdim);
	hipMemcpy(indata->data, d_imagePtr_cpy, xdim * ydim * zdim * sizeof(BYTE), hipMemcpyDeviceToHost);
	hipFree(d_imagePtr);
	hipFree(d_imagePtr_cpy);
	hipFree(d_compress);
	hipFree(d_decompress);
	hipFree(d_imagePtr_compact);
}

/*************** MAIN PROGRAM *****************/


int main(int argc, const char **argv)
{
	int i;
	int infile = -1;
	int outfile = -1;
	int gammavalue = -1;
	int xdim, ydim, zdim;
	BYTE max;

	TimerClock timer;

	{


		/* Parse command line params */
		for (i = 1; i < argc; i++)
		{
			if (strcmp(argv[i], "--help") == 0) {
				printf("\nUsage: %s INFILE [-g gamma] -o OUTFILE\n", argv[0]);
				printf("INFILE is the VTK file (unsigned char ()) to use as input.\n");
				printf("gamma is a value for the pruning parameter (default=1)\n");
				printf("gamma>1: constant pruning; gamma<1: linear pruning; gamma=0: square-root pruning.\n");
				return 0;
			}
			else if (strcmp(argv[i], "-g") == 0) {
				if (i + 1 < argc) {
					gammavalue = i + 1;
					i++;
				}
				else printf("Missing value for gamma.\n");
			}
			else if (strcmp(argv[i], "-o") == 0) {
				if (i + 1 < argc) {
					outfile = i + 1;
					i++;
				}
				else printf("Missing value for output file name.\n");
			}
			else infile = i;
		}

		if (infile == -1) {
			printf("Missing input file_name. Use '%s --help' for full help.\n", argv[0]);
			return 0;
		}
		if (outfile == -1) {
			printf("Missing output file_name. Use '%s --help' for full help.\n", argv[0]);
			return 0;
		}

		if (gammavalue != -1)
			gamma_val = atof(argv[gammavalue]);

		strcpy(input_file, argv[infile]);

		strcpy(output_file, argv[outfile]);

		basefilename = basename_no_ext(input_file, "vtk");

		sprintf(skel_file, "%s_%s%d_%s%s", basefilename, "g=", gamma_val, "skel", ".vtk");
		fprintf(stdout, "gamma = %d\n", gamma_val);
		fflush(stdout);

	}

	std::cerr << "Begin " << std::endl << std::endl;
	timer.update();

	auto size = new int[3];
	unsigned char* h_inputImagePtr = loadImage(input_file, size);
	xdim = size[0];
	ydim = size[1];
	zdim = size[2];
	int allsize = xdim * ydim * zdim;

	std::cerr << "Load cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	GetSkeleton(h_inputImagePtr, xdim, ydim, zdim);

	std::cerr << "Feature Trans cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	saveTiff(output_file, indata->data, size);

	// Cleanup
	ByteVolume_Delete(indata);

	return 0;
}
