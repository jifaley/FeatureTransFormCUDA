#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <string>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "volume.h"
#include "loadTiff.h"
#include "TimerClock.hpp"
#include "compaction.h"
#include "fastmarching.h"
#include "threshold.h"

#include <hip/hip_runtime.h> 
#include <>

const int GAMMA = 1;
const int FOREGROUND = 1;
const int BACKGROUND = 0;
const int SKEL = 255;

#define sqr(x) ((x)*(x))
#define sqrlength(i, j, k) ((i)*(i)+(j)*(j)+(k)*(k))

static int			gamma_val = GAMMA;
static IntVolume*	ft,*ft_test;
static ByteVolume*	indata, *indata_test;
static char			input_file[MAXSTR];
static char			output_file[MAXSTR];
static char*		basefilename;
static char			basename[MAXSTR];
static char			skel_file[MAXSTR];

__global__ void skeletonKernel(BYTE* skel, BYTE* dis, int xdim, int ydim, int zdim) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x <= 0 || x >= xdim - 1 || y <= 0 || y >= ydim - 1 || z <= 0 || z >= zdim - 1)
		return;

	int idx = x + y * xdim + z * xdim * ydim;

	if (dis[idx] == 0)
		return;

	int neighborOffsets[26][3] = {
		{-1, -1, -1}, { 0, -1, -1}, { 1, -1, -1},
		{-1,  0, -1}, { 0,  0, -1}, { 1,  0, -1},
		{-1,  1, -1}, { 0,  1, -1}, { 1,  1, -1},

		{-1, -1,  0}, { 0, -1,  0}, { 1, -1,  0},
		{-1,  0,  0},               { 1,  0,  0},
		{-1,  1,  0}, { 0,  1,  0}, { 1,  1,  0},

		{-1, -1,  1}, { 0, -1,  1}, { 1, -1,  1},
		{-1,  0,  1}, { 0,  0,  1}, { 1,  0,  1},
		{-1,  1,  1}, { 0,  1,  1}, { 1,  1,  1},
	};

	BYTE maxNeighborValue = 0;
	for (int i = 0; i < 26; i++) {
		int nx = x + neighborOffsets[i][0];
		int ny = y + neighborOffsets[i][1];
		int nz = z + neighborOffsets[i][2];

		if (nx >= 0 && nx < xdim && ny >= 0 && ny < ydim && nz >= 0 && nz < zdim) {
			if (nx <= x && ny <= y && nz <= z) {
				int neighborIdx = nx + ny * xdim + nz * xdim * ydim;
				maxNeighborValue = fmaxf(maxNeighborValue, dis[neighborIdx]);
			}
		}
	}

	if (dis[idx] < maxNeighborValue) {
		skel[idx] = SKEL;
	}
}

__global__ void markSimpleVoxels(BYTE* dis, int xdim, int ydim, int zdim) {
	// 当前线程的 3D 坐标
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;

	// 检查是否在图像范围内
	if (x <= 0 || x >= xdim - 1 || y <= 0 || y >= ydim - 1 || z <= 0 || z >= zdim - 1)
		return;

	// 当前体素的线性索引
	int idx = x + y * xdim + z * xdim * ydim;

	// 如果当前体素是背景，直接退出
	if (dis[idx] == 0)
		return;

	// 邻域偏移
	int neighborOffsets[26][3] = {
		{-1, -1, -1}, { 0, -1, -1}, { 1, -1, -1},
		{-1,  0, -1}, { 0,  0, -1}, { 1,  0, -1},
		{-1,  1, -1}, { 0,  1, -1}, { 1,  1, -1},

		{-1, -1,  0}, { 0, -1,  0}, { 1, -1,  0},
		{-1,  0,  0},               { 1,  0,  0},
		{-1,  1,  0}, { 0,  1,  0}, { 1,  1,  0},

		{-1, -1,  1}, { 0, -1,  1}, { 1, -1,  1},
		{-1,  0,  1}, { 0,  0,  1}, { 1,  0,  1},
		{-1,  1,  1}, { 0,  1,  1}, { 1,  1,  1},
	};

	// 记录邻域访问情况
	bool visited[26] = { false };
	int connectedComponents = 0;

	// 遍历 26 邻域进行连通性检查
	for (int i = 0; i < 26; i++) {
		int nx = x + neighborOffsets[i][0];
		int ny = y + neighborOffsets[i][1];
		int nz = z + neighborOffsets[i][2];

		if (nx >= 0 && nx < xdim && ny >= 0 && ny < ydim && nz >= 0 && nz < zdim) {
			int neighborIdx = nx + ny * xdim + nz * xdim * ydim;

			if (dis[neighborIdx] > 0 && !visited[i]) {
				connectedComponents++;

				for (int j = 0; j < 26; j++) {
					int nnx = x + neighborOffsets[j][0];
					int nny = y + neighborOffsets[j][1];
					int nnz = z + neighborOffsets[j][2];

					if (nnx >= nx - 1 && nnx <= nx + 1 &&
						nny >= ny - 1 && nny <= ny + 1 &&
						nnz >= nz - 1 && nnz <= nz + 1) {
						int nNeighborIdx = nnz * xdim * ydim + nny * xdim + nnx;
						if (dis[nNeighborIdx] > 0)
							visited[j] = true;
					}
				}
			}
		}
	}
	if (connectedComponents == 1) {
		dis[idx] = 0;
		return;
	}
}

__global__ void markSkeletonVoxels(BYTE* skel, int xdim, int ydim, int zdim) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x <= 0 || x >= xdim - 1 || y <= 0 || y >= ydim - 1 || z <= 0 || z >= zdim - 1)
		return;

	int idx = x + y * xdim + z * xdim * ydim;

	if (skel[idx] != 0) {
		skel[idx] = 255; // 将未被删除的体素标记为骨架
	}
}



__global__ void AddNeighbor(BYTE* skel, int xdim, int ydim, int zdim) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x <= 0 || x >= xdim - 1 || y <= 0 || y >= ydim - 1 || z <= 0 || z >= zdim - 1)
		return;

	int idx = x + y * xdim + z * xdim * ydim;

	int neighborOffsets[26][3] = {
		{-1, -1, -1}, { 0, -1, -1}, { 1, -1, -1},
		{-1,  0, -1}, { 0,  0, -1}, { 1,  0, -1},
		{-1,  1, -1}, { 0,  1, -1}, { 1,  1, -1},

		{-1, -1,  0}, { 0, -1,  0}, { 1, -1,  0},
		{-1,  0,  0},               { 1,  0,  0},
		{-1,  1,  0}, { 0,  1,  0}, { 1,  1,  0},

		{-1, -1,  1}, { 0, -1,  1}, { 1, -1,  1},
		{-1,  0,  1}, { 0,  0,  1}, { 1,  0,  1},
		{-1,  1,  1}, { 0,  1,  1}, { 1,  1,  1},
	};

	for (int i = 0; i < 26; i++) {
		int nx = x + neighborOffsets[i][0];
		int ny = y + neighborOffsets[i][1];
		int nz = z + neighborOffsets[i][2];

		if (nx >= 0 && nx < xdim && ny >= 0 && ny < ydim && nz >= 0 && nz < zdim) {

			int neighborIdx = nx + ny * xdim + nz * xdim * ydim;
			if (skel[neighborIdx] == SKEL)
				skel[idx] = SKEL;

		}
	}
}


__global__ void processArrayKernel(unsigned char* data, int size, int FOREGROUND, int SKEL) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) {
		if (data[idx] == FOREGROUND) {
			data[idx] = 0;
		}
		if (data[idx] == SKEL) {
			data[idx] = 1;
		}
	}
}

__global__ void meanBlurKernel(BYTE* input, BYTE* output, int xdim, int ydim, int zdim, int kernelRadius) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x < kernelRadius || x >= xdim - kernelRadius ||
		y < kernelRadius || y >= ydim - kernelRadius ||
		z < kernelRadius || z >= zdim - kernelRadius) {
		return;
	}

	int idx = x + y * xdim + z * xdim * ydim;

	float sum = 0.0f;
	int count = 0;

	for (int dz = -kernelRadius; dz <= kernelRadius; dz++) {
		for (int dy = -kernelRadius; dy <= kernelRadius; dy++) {
			for (int dx = -kernelRadius; dx <= kernelRadius; dx++) {
				int nx = x + dx;
				int ny = y + dy;
				int nz = z + dz;

				int neighborIdx = nx + ny * xdim + nz * xdim * ydim;

				sum += input[neighborIdx];
				count++;
			}
		}
	}

	output[idx] = (BYTE)(sum / count);
}

void GetSkeleton(unsigned char* ImagePtr, int xdim, int ydim, int zdim)
{
	TimerClock timer;

	unsigned char* d_imagePtr_ori = nullptr;
	hipMalloc(&d_imagePtr_ori, sizeof(unsigned char) * xdim * ydim * zdim);
	hipMemcpy(d_imagePtr_ori, ImagePtr, sizeof(unsigned char) * xdim * ydim * zdim, hipMemcpyHostToDevice);

	unsigned char* d_imagePtr_cpy = nullptr;
	hipMalloc(&d_imagePtr_cpy, sizeof(unsigned char) * xdim * ydim * zdim);
	hipMemcpy(d_imagePtr_cpy, ImagePtr, sizeof(unsigned char) * xdim * ydim * zdim, hipMemcpyHostToDevice);

	int globalThreshold = 35;
	globalThreshold = 2;
	addGlobalThreshold(d_imagePtr_ori, xdim, ydim, zdim, globalThreshold);

	unsigned char* d_imagePtr = nullptr;
	hipMalloc(&d_imagePtr, sizeof(unsigned char) * xdim * ydim * zdim);

	dim3 blockSize1(8, 8, 8);
	dim3 gridSize1((xdim + blockSize1.x - 1) / blockSize1.x,
		(ydim + blockSize1.y - 1) / blockSize1.y,
		(zdim + blockSize1.z - 1) / blockSize1.z);

	addMaxMinGlobalThreshold(d_imagePtr_cpy, xdim, ydim, zdim, globalThreshold);
	std::cerr << "set globalThreshold cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	/*
	//将原图中紧靠前景点的背景点赋值为1，作为之后的扩展起点
	addDarkPadding(d_imagePtr, xdim, ydim, zdim, globalThreshold);
	std::cerr << "add darkpadding cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	int* d_compress; //压缩映射矩阵
	int* d_decompress; //解压缩映射矩阵
	unsigned char* d_imagePtr_compact; //压缩后原图
	int newSize; //压缩后数组总大小

	compactImage(d_imagePtr, d_imagePtr_compact, d_compress, d_decompress, xdim, ydim, zdim, newSize);

	std::cerr << "OldSize: " << xdim * ydim * zdim << " NewSize: " << newSize << std::endl;
	printf("Compress Ratio: %.2lf%%\n", 100.0 * newSize / (1.0 * xdim * ydim * zdim));

	hipDeviceSynchronize();
	std::cerr << "compaction cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	//测试1: 直接将原图变换为距离值，调用addGreyWeightTransform() 函数 将d_imagePtr_compact 变为距离变换之后的数组

	addGreyWeightTransform(d_imagePtr, d_imagePtr_compact, d_compress, d_decompress, xdim, ydim, zdim, newSize);
	hipDeviceSynchronize();
	std::cerr << "GreyWeight Transform cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	//调用recoverImage() 将d_imagePtr_compact中的数据解压缩到原图d_imagePtr中
	recoverImage(d_imagePtr, d_imagePtr_compact, d_decompress, newSize);

	//skeletonKernel << <gridSize1, blockSize1 >> > (d_imagePtr_cpy, d_imagePtr, xdim, ydim, zdim);*/

	unsigned char* d_host = (unsigned char*)malloc(sizeof(unsigned char) * xdim * ydim * zdim);
	for (int i = 0; i < 1; i++) {
		markSimpleVoxels << <gridSize1, blockSize1 >> > (d_imagePtr_cpy, xdim, ydim, zdim);
	}
	markSkeletonVoxels << <gridSize1, blockSize1 >> > (d_imagePtr_cpy, xdim, ydim, zdim);

	int blockSize2 = 256;
	int numBlocks = (xdim * ydim * zdim + 256 - 1) / 256;

	processArrayKernel << <numBlocks, blockSize2 >> > (d_imagePtr_cpy, xdim * ydim * zdim, FOREGROUND, SKEL);

	indata = ByteVolume_New(xdim, ydim, zdim);
	hipMemcpy(indata->data, d_imagePtr_cpy, xdim * ydim * zdim * sizeof(BYTE), hipMemcpyDeviceToHost);
	hipFree(d_imagePtr);
	hipFree(d_imagePtr_cpy);
}

/*************** MAIN PROGRAM *****************/


int main(int argc, const char **argv)
{
	int i;
	int infile = -1;
	int outfile = -1;
	int gammavalue = -1;
	int xdim, ydim, zdim;
	BYTE max;

	TimerClock timer;

	{


		/* Parse command line params */
		for (i = 1; i < argc; i++)
		{
			if (strcmp(argv[i], "--help") == 0) {
				printf("\nUsage: %s INFILE [-g gamma] -o OUTFILE\n", argv[0]);
				printf("INFILE is the VTK file (unsigned char ()) to use as input.\n");
				printf("gamma is a value for the pruning parameter (default=1)\n");
				printf("gamma>1: constant pruning; gamma<1: linear pruning; gamma=0: square-root pruning.\n");
				return 0;
			}
			else if (strcmp(argv[i], "-g") == 0) {
				if (i + 1 < argc) {
					gammavalue = i + 1;
					i++;
				}
				else printf("Missing value for gamma.\n");
			}
			else if (strcmp(argv[i], "-o") == 0) {
				if (i + 1 < argc) {
					outfile = i + 1;
					i++;
				}
				else printf("Missing value for output file name.\n");
			}
			else infile = i;
		}

		if (infile == -1) {
			printf("Missing input file_name. Use '%s --help' for full help.\n", argv[0]);
			return 0;
		}
		if (outfile == -1) {
			printf("Missing output file_name. Use '%s --help' for full help.\n", argv[0]);
			return 0;
		}

		if (gammavalue != -1)
			gamma_val = atof(argv[gammavalue]);

		strcpy(input_file, argv[infile]);

		strcpy(output_file, argv[outfile]);

		basefilename = basename_no_ext(input_file, "vtk");

		sprintf(skel_file, "%s_%s%d_%s%s", basefilename, "g=", gamma_val, "skel", ".vtk");
		fprintf(stdout, "gamma = %d\n", gamma_val);
		fflush(stdout);

	}

	std::cerr << "Begin " << std::endl << std::endl;
	timer.update();

	auto size = new int[3];
	unsigned char* h_inputImagePtr = loadImage(input_file, size);
	xdim = size[0];
	ydim = size[1];
	zdim = size[2];
	int allsize = xdim * ydim * zdim;

	std::cerr << "Load cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	GetSkeleton(h_inputImagePtr, xdim, ydim, zdim);

	std::cerr << "Feature Trans cost " << timer.getTimerMilliSec() << "ms" << std::endl << std::endl;
	timer.update();

	saveTiff(output_file, indata->data, size);

	// Cleanup
	ByteVolume_Delete(indata);

	return 0;
}
